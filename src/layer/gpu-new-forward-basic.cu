#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define cudaErrChk(stmt) \
  { cudaAssert((stmt), __FILE__, __LINE__); }

inline void cudaAssert(hipError_t error,
                       const char* file,
                       int line,
                       bool abort = true) {
  if (error != hipSuccess) {
    std::cerr << "CUDA error: "
              << hipGetErrorString(error) << ' ' << file << ':' << line << std::endl;
    if (abort) {
      exit(error);
    }
  }
}

#define TILE_WIDTH 16

__global__ void conv_forward_kernel(float *y, 
                                    const float *x, 
                                    const float *k, 
                                    const int B, 
                                    const int M, 
                                    const int C, 
                                    const int H, 
                                    const int W, 
                                    const int K)
{
	// Calculate output dimensions
    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    // An example use of these macros:
    // float a = y4d(0,0,0,0)
    // y4d(0,0,0,0) = a
#define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define x4d(i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define k4d(i3, i2, i1, i0) k[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]


	// Calculate grid dimensions for parallelization
    int H_grid = ceil(1.0*H_out / TILE_WIDTH);
    int W_grid = ceil(1.0*W_out / TILE_WIDTH); 
    
	// Extract thread indices
    int b = blockIdx.x;                 // batch number
    int m = blockIdx.y;                 // output feature map
    int h = (blockIdx.z / W_grid) * TILE_WIDTH + threadIdx.y; // row of image matrix
    int w = (blockIdx.z % W_grid) * TILE_WIDTH + threadIdx.x; // col of image matrix
    
	// Param for convolution result
    float accum = 0.0f;
	// Perform convolution by summing over input features and applying the filter
    if ((h < (H_out)) && (w < (W_out))) 
    {
        float accum = 0.0f;
        for(int c=0; c<C; c++)             // sum over all input features
        {
            for(int p=0; p<K; p++)         // KxK filter 
                for(int q=0; q<K; q++)
                    accum += x4d(b, c, h+p, w+q) * k4d(m, c, p, q); // 4 dimensions macro resolve thread index
        }
		// Store the result in the output array
        y4d(b,m,h,w) = accum;
    }
	// Undefine macros to avoid potential conflicts
#undef y4d
#undef x4d
#undef k4d
}
	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_y, 
                                                    const float *host_x, 
                                                    const float *host_k, 
                                                    float **device_y_ptr, 
                                                    float **device_x_ptr, 
                                                    float **device_k_ptr, 
                                                    const int B, 
                                                    const int M, 
                                                    const int C, 
                                                    const int H, 
                                                    const int W, 
                                                    const int K)
{
    // Allocate memory and copy data to GPU(device)
    printf("(B=%d, M=%d, C=%d, H=%d, W=%d, K=%d)\n", B, M, C, H, W, K);

    // We pass double pointers for you to initialize the relevant device pointers,
    // which are passed to the other two functions.

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    const int inputSize  = B * C * H * W * sizeof(float);  // input features map C
    const int outputSize = B * M * H_out * W_out * sizeof(float); // output feature map M
    const int filterSize = M * C * K * K * sizeof(float); // C * M filter Maps of size K*K

    hipMalloc((void **) device_x_ptr, inputSize);
    hipMalloc((void **) device_y_ptr, outputSize);
    hipMalloc((void **) device_k_ptr, filterSize);

    // Copy input data to device
    hipMemcpy(*device_x_ptr, host_x, inputSize, hipMemcpyHostToDevice);
    // Copy filter size to device
    hipMemcpy(*device_k_ptr, host_k, filterSize, hipMemcpyHostToDevice);

}


__host__ void GPUInterface::conv_forward_gpu(float *device_y, 
                                             const float *device_x, 
                                             const float *device_k, 
                                             const int B, 
                                             const int M, 
                                             const int C, 
                                             const int H, 
                                             const int W, 
                                             const int K)
{
    // Set the kernel dimensions and call the kernel

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    const int H_grid = ceil(1.0*H_out / TILE_WIDTH);
    int W_grid = ceil(1.0*W_out / TILE_WIDTH);
    int Z = H_grid * W_grid;

    // Block dimensions = #of threads in the block
    dim3 numThreadsPerBlock(TILE_WIDTH, TILE_WIDTH, 1);

    // Grid Dimension = #of Blocks: Batch Size(B) * Number of Output Features(M) * Calculated gridsize(Z)
    dim3 numBlocksInGrid(B, M, Z);


    // launch the kernel
    conv_forward_kernel<<<numBlocksInGrid, numThreadsPerBlock>>>(device_y, device_x, device_k, B, M, C, H, W, K);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_y, 
                                                    float *device_y, 
                                                    float *device_x, 
                                                    float *device_k, 
                                                    const int B, 
                                                    const int M, 
                                                    const int C, 
                                                    const int H, 
                                                    const int W, 
                                                    const int K)
{
    // Copy the output from device back to host
    
    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    int outputSize = B * M * H_out * W_out * sizeof(float);

    hipMemcpy(host_y, device_y, outputSize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_k);
}

