#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <mma.h>
#include "gpu-new-forward.h"

#define cudaErrChk(stmt) \
  { cudaAssert((stmt), __FILE__, __LINE__); }

inline void cudaAssert(hipError_t error,
                       const char* file,
                       int line,
                       bool abort = true) {
  if (error != hipSuccess) {
    std::cerr << "CUDA error: "
              << hipGetErrorString(error) << ' ' << file << ':' << line << std::endl;
    if (abort) {
      exit(error);
    }
  }
}

// Some feature flags
//#define USE_STREAM // Use multi-stream to accelerate transfers
//#define USE_ASYNC_ALLOCATOR // Use async allocators, available >= 11.2

// Allocate maximal possible kernel size and reuse it between op1/2
#define M_MAX 16
#define C_MAX 4
#define KERNEL_WIDTH 7
__constant__ float kernel[M_MAX * C_MAX * KERNEL_WIDTH * KERNEL_WIDTH];

// The actual convolution kernel
#define TILE_WIDTH 16
#define PADDED_TILE_WIDTH (TILE_WIDTH + KERNEL_WIDTH - 1)
__global__ void conv_forward_kernel_op(float* __restrict__ y,
                                    const float* __restrict__ x,
                                    const int B,
                                    const int M,
                                    const int C,
                                    const int H,
                                    const int W,
                                    const int K) {
  extern __shared__ float tile[];

  /*
  Function paramter definitions:
  y - output
  x - input
  k - kernel
  B - batch_size (number of images in x)
  M - number of output feature maps
  C - number of input feature maps
  H - input height dimension
  W - input width dimension
  K - kernel height and width (K x K)
  */

  const int H_out = H - K + 1;
  const int W_out = W - K + 1;

#define y2d(i1, i0) \
  y[b * (M * H_out * W_out) + m * (H_out * W_out) + (i1) * (W_out) + i0]
#define t3d(i2, i1, i0)                                   \
  tile[tb * (C * PADDED_TILE_WIDTH * PADDED_TILE_WIDTH) + \
       (i2) * (PADDED_TILE_WIDTH * PADDED_TILE_WIDTH) +   \
       (i1) * (PADDED_TILE_WIDTH) + i0]
#define x3d(i2, i1, i0) \
  x[b * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define k3d(i2, i1, i0) \
  kernel[m * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

  // Alias for block/thread index
  const int bx = blockIdx.x, by = blockIdx.y;
  const int tx = threadIdx.x, ty = threadIdx.y;
  // Alias for batch axis
  const int tb = threadIdx.z;
  const int b = blockIdx.z * blockDim.z + tb;

  int dst_x, dst_y, src_x, src_y;

  for (int m = 0; m < M; m++) {
    // Pre-load to shared memory, need to loop multiple time, PW^2 / W^2
    for (int c = 0; c < C; c++) {
      for (int dst = ty * TILE_WIDTH + tx;
           dst < PADDED_TILE_WIDTH * PADDED_TILE_WIDTH;
           dst += TILE_WIDTH * TILE_WIDTH) {
        // 3D index inside a padded tiles
        dst_x = dst % PADDED_TILE_WIDTH;
        dst_y = dst / PADDED_TILE_WIDTH;
        // 3D index in global array, simply subtract the pad size
        src_x = (bx * TILE_WIDTH + dst_x);
        src_y = (by * TILE_WIDTH + dst_y);

        if ((src_x < W) && (src_y < H)) {
          t3d(c, dst_y, dst_x) = x3d(c, src_y, src_x);
        } else {
          t3d(c, dst_y, dst_x) = 0.0f;
        }
      }
    }
    __syncthreads();

    // the actual convolution
    float sum = 0;
    for (int c = 0; c < C; c++) {
#pragma unroll
      for (int p = 0; p < KERNEL_WIDTH; p++) {
#pragma unroll
        for (int q = 0; q < KERNEL_WIDTH; q++) {
          sum += t3d(c, ty + p, tx + q) * k3d(c, p, q);
        }
      }
    }

    // update the destination 3D index
    dst_x = bx * TILE_WIDTH + tx;
    dst_y = by * TILE_WIDTH + ty;
    // restore the linear index in global scope
    if ((dst_x < W_out) && (dst_y < H_out)) {
      y2d(dst_y, dst_x) = sum;
    }
    __syncthreads();
  }

#undef y2d
#undef t3d
#undef x3d
#undef k3d
}

__host__ void GPUInterface2::conv_forward_gpu_prolog(const float* host_y,
                                                    const float* host_x,
                                                    const float* host_k,
                                                    float** device_y_ptr,
                                                    float** device_x_ptr,
                                                    float** device_k_ptr,
                                                    const int B,
                                                    const int M,
                                                    const int C,
                                                    const int H,
                                                    const int W,
                                                    const int K) {
  std::cout << "*** constant mem + tiled + restrict/unroll + stream ***" << std::endl;

  printf("(B=%d, M=%d, C=%d, H=%d, W=%d, K=%d)\n", B, M, C, H, W, K);

  // Estimat output dimension
  const int H_out = H - K + 1;
  const int W_out = W - K + 1;

  // Calculate needed bytes
  const size_t bytes_y = (B * M * H_out * W_out) * sizeof(float);
  const size_t bytes_x = (B * C * H * W) * sizeof(float);
  const size_t bytes_k = (M * C * K * K) * sizeof(float);

#ifndef USE_STREAM
  // Allocate memory on device
  cudaErrChk(hipMalloc(device_y_ptr, bytes_y));
  cudaErrChk(hipMalloc(device_x_ptr, bytes_x));

  // Copy input data to device
  cudaErrChk(hipMemcpy(*device_x_ptr, host_x, bytes_x, hipMemcpyHostToDevice));
#else
  // Pass through host pointers
  *device_y_ptr = (float*)host_y;
  *device_x_ptr = (float*)host_x;

  // Mark them as pinned memory for asynchronous transfer
  cudaErrChk(hipHostRegister(*device_y_ptr, bytes_y, hipHostRegisterPortable));
  cudaErrChk(hipHostRegister(*device_x_ptr, bytes_x, hipHostRegisterPortable));
#endif

  // Copy kernel weights
  cudaErrChk(hipMemcpyToSymbol(HIP_SYMBOL(kernel), host_k, bytes_k));
}

__host__ void GPUInterface2::conv_forward_gpu(float* device_y,
                                             const float* device_x,
                                             const float* device_k,  // unused
                                             const int B0,
                                             const int M,
                                             const int C,
                                             const int H,
                                             const int W,
                                             const int K) {
  // Estimat output dimension
  const int H_out = H - K + 1;
  const int W_out = W - K + 1;

  // Block size along the B (batch) dimension
  const int B_batch_size = 4;

  /*** Prolog BEGIN ***/
#ifndef USE_STREAM
  // Send the entire batch
  const int B = B0;
#else // USE_STREAM
  // Create streams
  const int n_streams = 16;
  hipStream_t stream[n_streams];
  for (int i = 0; i < n_streams; i++) {
    cudaErrChk(hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking));
  }

  // We pass through host pointers from the prolog function
  const float* host_x = device_x;
  float* host_y = device_y;

  // Calculate total elements and bytes
  const int n_y = B0 * M * H_out * W_out;
  const int n_x = B0 * C * H * W;
  const size_t bytes_y = n_y * sizeof(float);
  const size_t bytes_x = n_x * sizeof(float);
  // Calculate partial elements and bytes per stream
  const int B = ceil((float)B0 / n_streams);
  const int n_y_stream = B * M * H_out * W_out;
  const int n_x_stream = B * C * H * W;

#ifndef USE_ASYNC_ALLOCATOR
  cudaErrChk(hipMalloc(&device_y, bytes_y));
  cudaErrChk(hipMalloc(&device_x, bytes_x));

  // Copy over the relevant data structures to the GPU
  for (int i = 0; i < n_streams; i++) {
    size_t offset = i * n_x_stream;
    size_t bytes = n_x_stream * sizeof(float);
    if (offset + n_x_stream > n_x) {
      // Last stream does not need to copy that much
      bytes = (n_x - offset) * sizeof(float);
    }
    cudaErrChk(hipMemcpyAsync((void*)&device_x[offset], (void*)&host_x[offset], bytes,
                               hipMemcpyHostToDevice, stream[i]));
  }
#else // USE_ASYNC_ALLOCATOR
  // Allocate memory input segments of each stream
  float* device_y_stream[n_streams];
  float* device_x_stream[n_streams];
  for (int i = 0; i < n_streams; i++) {
    size_t offset = i * n_x_stream;
    size_t bytes = n_x_stream * sizeof(float);
    if (offset + n_x_stream > n_x) {
      // Last stream does not need to copy that much
      bytes = (n_x - offset) * sizeof(float);
    }
    cudaErrChk(hipMallocAsync(&device_x_stream[i], bytes, stream[i]));

    // Copy over that chunk from host
    cudaErrChk(hipMemcpyAsync(&device_x_stream[i], &host_x[offset], bytes,
                               hipMemcpyHostToDevice, stream[i]));
  }

  // Allocate memory for output segments
  for (int i = 0; i < n_streams; i++) {
    size_t offset = i * n_y_stream;
    size_t bytes = n_y_stream * sizeof(float);
    if (offset + n_y_stream > n_y) {
      // Last stream does not need to copy that much
      bytes = (n_y - offset) * sizeof(float);
    }
    cudaErrChk(hipMallocAsync(&device_y_stream[i], bytes, stream[i]));
  }
#endif // USE_ASYNC_ALLOCATOR
#endif // USE_STREAM
  /*** Prolog END ***/


  /*** Kernel call BEGIN ***/
  // Calculate launch size
  dim3 block(TILE_WIDTH, TILE_WIDTH, 1);
  dim3 grid(ceil((float)W_out / block.x),
            ceil((float)H_out / block.y),
            ceil((float)B / block.z));
  printf("grid=(x=%d, y=%d, z=%d), block=(x=%d, y=%d, z=%d)\n",
         grid.x, grid.y, grid.z, block.x, block.y, block.z);

  // Determine shared memory size
  size_t smem_size =
      B_batch_size * C * PADDED_TILE_WIDTH * PADDED_TILE_WIDTH * sizeof(float);

  // Call the kernel
#ifndef USE_STREAM
  conv_forward_kernel_op<<<grid, block, smem_size>>>(device_y, device_x, B, M, C, H, W, K);
  cudaErrChk(hipDeviceSynchronize());
#else // USE_STREAM
  for (int i = 0; i < n_streams; i++) {
#ifndef USE_ASYNC_ALLOCATOR
    size_t offset_y = i * n_y_stream;
    size_t offset_x = i * n_x_stream;
    conv_forward_kernel_op<<<grid, block, smem_size, stream[i]>>>(
        &device_y[offset_y], &device_x[offset_x], B, M, C, H, W, K);
#else // USE_ASYNC_ALLOCATOR
    conv_forward_kernel_op<<<grid, block, smem_size, stream[i]>>>(
        device_y_stream[i], device_x_stream[i], B, M, C, H, W, K);
#endif // USE_ASYNC_ALLOCATOR
  }
#endif // USE_STREAM
  /*** Kernel call END ***/


  /*** Epilog BEGIN ***/
#ifndef USE_STREAM
  // We directly wait for the single kernel to end
  cudaErrChk(hipDeviceSynchronize());
#else
  // Copy back data to host
  for (int i = 0; i < n_streams; i++) {
    size_t offset = i * n_y_stream;
    size_t bytes = n_y_stream * sizeof(float);
    if (offset + n_y_stream > n_y) {
      // Last stream does not need to copy that much
      bytes = (n_y - offset) * sizeof(float);
    }
#ifndef USE_ASYNC_ALLOCATOR
    cudaErrChk(hipMemcpyAsync(&host_y[offset], &device_y[offset], bytes,
                               hipMemcpyDeviceToHost, stream[i]));
#else // USE_ASYNC_ALLOCATOR
    cudaErrChk(hipMemcpyAsync(&host_y[offset], device_y_stream[i], bytes,
                               hipMemcpyDeviceToHost, stream[i]));

    // Free segments for current stream
    cudaErrChk(hipFreeAsync(device_y_stream[i], stream[i]));
    cudaErrChk(hipFreeAsync(device_x_stream[i], stream[i]));
#endif // USE_ASYNC_ALLOCATOR
  }

  // Need to wait every stream to finish before destory all streams
  cudaErrChk(hipDeviceSynchronize());

  // Destory streams
  for (int i = 0; i < n_streams; i++) {
    cudaErrChk(hipStreamDestroy(stream[i]));
  }
#endif
  /*** Epilog END ***/
}

__host__ void GPUInterface2::conv_forward_gpu_epilog(float* host_y,
                                                    float* device_y,
                                                    float* device_x,
                                                    float* device_k,
                                                    const int B,
                                                    const int M,
                                                    const int C,
                                                    const int H,
                                                    const int W,
                                                    const int K) {
#ifndef USE_STREAM
  const int H_out = H - K + 1;
  const int W_out = W - K + 1;
  const size_t bytes_y = (B * M * H_out * W_out) * sizeof(float);

  // Copy output back to host
  cudaErrChk(hipMemcpy(host_y, device_y, bytes_y, hipMemcpyDeviceToHost));

  // Free device memory
  cudaErrChk(hipFree(device_y));
  cudaErrChk(hipFree(device_x));
#else
  // Data is already write back to host earlier, safe to clean up now

  // Release pinned memory
  cudaErrChk(hipHostUnregister(device_y));
  cudaErrChk(hipHostUnregister(device_x));
#endif
}

// __host__ void GPUInterface::get_device_properties() {
//   int deviceCount;
//   hipGetDeviceCount(&deviceCount);

//   for (int dev = 0; dev < deviceCount; dev++) {
//     hipDeviceProp_t deviceProp;
//     hipGetDeviceProperties(&deviceProp, dev);

//     std::cout << "Device " << dev << " name: " << deviceProp.name << std::endl;
//     std::cout << "Computational capabilities: "
//               << deviceProp.major << "." << deviceProp.minor << std::endl;
//     std::cout << "Max Global memory size: " << deviceProp.totalGlobalMem
//               << std::endl;
//     std::cout << "Max Constant memory size: " << deviceProp.totalConstMem
//               << std::endl;
//     std::cout << "Max Shared memory size per block: " << deviceProp.sharedMemPerBlock
//               << std::endl;
//     std::cout << "Max threads per block: " << deviceProp.maxThreadsPerBlock
//               << std::endl;
//     std::cout << "Max block dimensions: "
//               << deviceProp.maxThreadsDim[0] << " x, "
//               << deviceProp.maxThreadsDim[1] << " y, "
//               << deviceProp.maxThreadsDim[2] << " z" << std::endl;
//     std::cout << "Max grid dimensions: "
//               << deviceProp.maxGridSize[0] << " x, "
//               << deviceProp.maxGridSize[1] << " y, "
//               << deviceProp.maxGridSize[2] << " z" << std::endl;
//     std::cout << "Warp Size: " << deviceProp.warpSize << std::endl;
//   }
// }
